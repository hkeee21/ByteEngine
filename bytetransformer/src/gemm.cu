#include "hip/hip_runtime.h"
// Copyright 2023 Bytedance Ltd. and/or its affiliates.
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */
#include "bytetransformer/include/gemm.h"

namespace bytetransformer {
void dense_layer_kernel_launcher(const float *in, const float *weight, float *out, const int M,
                                 const int K, const int N, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const float alpha = 1.0f, beta = 0.0f;
  check_cuda_error(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, weight,
                                HIP_R_32F, N, in, HIP_R_32F, K, &beta, out, HIP_R_32F, N,
                                HIP_R_32F, static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

void dense_layer_kernel_launcher(const __half *in, const __half *weight, __half *out, const int M,
                                 const int K, const int N, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const __half alpha = (__half)1.0f, beta = (__half)0.0f;
  check_cuda_error(hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, weight,
                                HIP_R_16F, N, in, HIP_R_16F, K, &beta, out, HIP_R_16F, N,
                                HIP_R_16F, static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

void cublas_Gemm_Strided_Batched(const float *A, const float *B, float *out, const int M,
                                 const int K, const int N, const int batch_count,
                                 hipblasOperation_t trans_A, hipblasOperation_t trans_B, float alpha,
                                 float beta, hipblasHandle_t cublas_handle, hipStream_t stream,
                                 int cublasAlgo) {
  const int lda = (trans_A == HIPBLAS_OP_N) ? K : M;
  const int ldb = (trans_B == HIPBLAS_OP_N) ? N : K;

  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, trans_B, trans_A, N, M, K, &alpha, B, HIP_R_32F, ldb, K * N, A, HIP_R_32F,
      lda, M * K, &beta, out, HIP_R_32F, N, M * N, batch_count, HIP_R_32F,
      static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}

void cublas_Gemm_Strided_Batched(const __half *A, const __half *B, __half *out, const int M,
                                 const int K, const int N, const int batch_count,
                                 hipblasOperation_t trans_A, hipblasOperation_t trans_B,
                                 __half alpha, __half beta, hipblasHandle_t cublas_handle,
                                 hipStream_t stream, int cublasAlgo) {
  const int lda = (trans_A == HIPBLAS_OP_N) ? K : M;
  const int ldb = (trans_B == HIPBLAS_OP_N) ? N : K;

  check_cuda_error(hipblasGemmStridedBatchedEx(
      cublas_handle, trans_B, trans_A, N, M, K, &alpha, B, HIP_R_16F, ldb, K * N, A, HIP_R_16F,
      lda, M * K, &beta, out, HIP_R_16F, N, M * N, batch_count, HIP_R_16F,
      static_cast<hipblasGemmAlgo_t>(cublasAlgo)));
}
}  // namespace bytetransformer
